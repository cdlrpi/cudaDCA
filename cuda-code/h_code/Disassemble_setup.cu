#include "hip/hip_runtime.h"
#include "classes.h"
#include <iostream>
__global__ void Disassemble(float Xinv[],float Zs[],float oldAF[], float newAF[], int numBlocks);
void printm(float A[6][6]);
void printa(float A[], int x);
void cudaDisassemble(float OldAF[], Body *morebds, Body *lessbds, int odd, int morelen, int lesslen, float AF[])
{	
	float *d_oldAF, *d_newAF,*d_Xinv, *zs, *newAF, *Xinv, *oldAF;
	float *d_zs;
	int r11,r12,r13,r21,r22,r23,rXinv;
	int newlen = (int) morelen*4;
	int gpu_len = (int) (newlen-(4*odd));
	int numBlocks = (int) morelen-lesslen;
	zs = (float*) malloc(sizeof(float)*(numBlocks*2)*6*26);
	newAF = (float*) malloc(sizeof(float)*(gpu_len)*6);
	oldAF = (float*) malloc(sizeof(float)*(lesslen-odd)*4*6);
	Xinv = (float*) malloc(sizeof(float)*(numBlocks)*5*5);

	
	for(int c = 0; c<(lesslen-odd)*4; c++)
	{
		for(int r = 0; r<6; r++)
		{
			oldAF[c+r*(lesslen-odd)*4]= OldAF[c+r*lesslen*4];
		}
	}
	for(int i = 0; i<numBlocks*2; i++)
	{
		for(int c = 0; c<6; c++)
		{
			for(int r = 0; r<6; r++)
			{	r11=26*(numBlocks*2)*r+c+i*26;
				r12=r11+6;
				r13=r12+6;
				r21=r13+1;
				r22=r21+6;
				r23=r22+6;
				
				zs[r11] = morebds[i].z11[r][c];
				zs[r12] = morebds[i].z12[r][c];
				zs[r21] = morebds[i].z21[r][c];
				zs[r22] = morebds[i].z22[r][c];
				
				if(c==0)
				{
					zs[r13] = morebds[i].z13[r];
					zs[r23] = morebds[i].z23[r];
				}
			}
		}
	}

	for(int i = 0; i<numBlocks; i++)
	{
		for(int c = 0; c<5; c++)
		{
			for(int r = 0; r<5; r++)
			{	
				rXinv = i*5+c+r*5*numBlocks;
				Xinv[rXinv]= lessbds[i].Xinv[r][c];
			}
		}
	}

	
   
	std::cout<<hipMalloc(&d_zs,sizeof(float)*(numBlocks*2)*6*26);
	std::cout<<hipDeviceSynchronize();
	std::cout<<hipMalloc(&d_newAF,sizeof(float)*(gpu_len)*6);
	std::cout<<hipDeviceSynchronize();
	std::cout<<hipMalloc(&d_oldAF,sizeof(float)*(lesslen-odd)*4*6);
	std::cout<<hipDeviceSynchronize();
	std::cout<<hipMalloc(&d_Xinv,sizeof(float)*(numBlocks)*5*5);	
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(d_zs, zs, sizeof(float)*(numBlocks*2)*6*26, hipMemcpyHostToDevice);
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(d_Xinv, Xinv, sizeof(float)*(numBlocks)*5*5, hipMemcpyHostToDevice);
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(d_oldAF, oldAF, sizeof(float)*(lesslen-odd)*4*6, hipMemcpyHostToDevice);

	dim3 dimBlock(6, 6,1);
	dim3 dimGrid(numBlocks,1,1);
	std::cout<<hipDeviceSynchronize();
	Disassemble<<<dimGrid, dimBlock>>>(d_Xinv, d_zs, d_oldAF, d_newAF, numBlocks);
	std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(newAF, d_newAF,sizeof(float)*(gpu_len)*6, hipMemcpyDeviceToHost);
	std::cout<<hipDeviceSynchronize();
	for(int c = 0; c<(morelen)*4; c++)
	{
		for (int r = 0; r<6;r++)
		{	
			
			AF[r*morelen*4+c] = newAF[c+r*(gpu_len)];
		
		}
	}
	if(odd ==1)
	{
		for (int r = 0; r<6;r++)
		{
			AF[r*morelen*4+morelen*4-4]=OldAF[r*lesslen*4+lesslen*4-4];
			AF[r*morelen*4+morelen*4-3]=OldAF[r*lesslen*4+lesslen*4-3];
			AF[r*morelen*4+morelen*4-2]=OldAF[r*lesslen*4+lesslen*4-2];
			AF[r*morelen*4+morelen*4-1]=OldAF[r*lesslen*4+lesslen*4-1];
		}
	}
	//std::cin.get();
	
	std::cout<<hipFree(d_zs);
	std::cout<<hipFree(d_newAF);
	std::cout<<hipFree(d_oldAF);
	std::cout<<hipFree(d_Xinv);
	
	free(zs);
	free(newAF);
	free(oldAF);
	free(Xinv);

}

	
