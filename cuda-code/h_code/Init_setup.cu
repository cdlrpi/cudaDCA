#include "hip/hip_runtime.h"
#include "classes.h"
__global__ void Initialize(float state[],float m[], float l[],float I[],float Zetas[],int n);
void printa(float a[], int n);
#include <iostream>
void CudaInitialize(InitBody* oldbds, float x[], int n, Body *newbds)
{
	int x_size = 2*n;
	int I_size = 3*n*3;
	int z_size = 26*6*n;
	int r11,r12,r13,r21,r22,r23;
	float *x_gpu= (float*)malloc(x_size*sizeof(float));
	float *m_gpu =(float*)malloc(n*sizeof(float));
	float *l_gpu = (float*)malloc(n*sizeof(float));
	float *I_gpu = (float*)malloc(I_size*sizeof(float));
	float *zs_gpu = (float*)malloc(z_size*sizeof(float));
	float *d_x, *d_m, *d_l, *d_I, *d_zs;
	
	for(int i = 0; i<n; i++)
	{
		x_gpu[i] = x[i];
		x_gpu[i+n] = x[i + n];
		m_gpu[i] = oldbds[i].m;
		l_gpu[i]= oldbds[i].l;
		for(int c = 0; c<3; c++)
		{
			for(int r = 0; r<3; r++)
			{
				I_gpu[3*n*r+c+i*3]=oldbds[i].I[r][c];
			}
		}
	}

	// Allocate and Load M and N to device memor

	std::cout<<hipMalloc(&d_x,x_size*sizeof(float));
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMalloc(&d_m, n*sizeof(float));
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMalloc(&d_l, n*sizeof(float));
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMalloc(&d_I, I_size*sizeof(float));
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMalloc(&d_zs, z_size*sizeof(float));
	std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(d_x, x_gpu, x_size*sizeof(float), hipMemcpyHostToDevice);
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(d_m, m_gpu, n*sizeof(float), hipMemcpyHostToDevice);
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(d_I, I_gpu, I_size*sizeof(float), hipMemcpyHostToDevice);
std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(d_l, l_gpu, n*sizeof(float), hipMemcpyHostToDevice);
	dim3 dimBlock(6, 6,1);
	dim3 dimGrid(n,1,1);
	hipDeviceSynchronize();
	
	Initialize<<<dimGrid, dimBlock>>>(d_x, d_m, d_l, d_I, d_zs, n);	
	std::cout<<hipDeviceSynchronize();
	std::cout<<hipMemcpy(zs_gpu, d_zs, z_size*sizeof(float), hipMemcpyDeviceToHost);
	std::cout<<hipDeviceSynchronize();
	for(int i = 0; i<n; i++)
	{
		for(int c = 0; c<6; c++)
		{
			for(int r = 0; r<6; r++)
			{	r11=26*n*r+c+i*26;
				r12=r11+6;
				r13=r12+6;
				r21=r13+1;
				r22=r21+6;
				r23=r22+6;
				newbds[i].z11[r][c]=zs_gpu[r11];
				newbds[i].z12[r][c]=zs_gpu[r12];
				newbds[i].z21[r][c]=zs_gpu[r21];
				newbds[i].z22[r][c]=zs_gpu[r22];
				if(c==0)
				{
					newbds[i].z13[r]=zs_gpu[r13];
					newbds[i].z23[r]=zs_gpu[r23];
				}
			}
			
		}

	
	}
	printa(newbds[2].z13,6);
	printa(newbds[2].z23,6);

	std::cout<<hipFree(d_x);
	std::cout<<hipFree(d_m);
	std::cout<<hipFree(d_l);
	std::cout<<hipFree(d_I);
	std::cout<<hipFree(d_zs);
	
free(x_gpu);
	free(m_gpu);
	free(l_gpu);
	free(I_gpu);
	free(zs_gpu);

}
