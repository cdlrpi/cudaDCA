#include "hip/hip_runtime.h"
#include "classes.h"
__global__ void Initialize(float state[],float m[], float l[],float I[],float Zetas[],int n);


void CudaInitialize(InitBody* oldbds, float x[], int n, Body *newbds)
{
	int x_size = 2*n;
	int I_size = 3*n*3;
	int z_size = 26*6*n;
	int r11,r12,r13,r21,r22,r23;
	float *x_gpu= (float*)malloc(x_size*sizeof(float));
	float *m_gpu =(float*)malloc(n*sizeof(float));
	float *l_gpu = (float*)malloc(n*sizeof(float));
	float *I_gpu = (float*)malloc(I_size*sizeof(float));
	float *zs_gpu = (float*)malloc(z_size*sizeof(float));
	float *d_x, *d_m, *d_l, *d_I, *d_zs;
	
	for(int i = 0; i<n; i++)
	{
		x_gpu[i] = x[i];
		x_gpu[i+n] = x[i + n];
		m_gpu[i] = oldbds[i].m;
		l_gpu[i]= oldbds[i].l;
		for(int c = 0; c<3; c++)
		{
			for(int r = 0; r<3; r++)
			{
				I_gpu[3*n*r+c+i*3]=oldbds[i].I[r][c];
			}
		}
	}
	// Allocate and Load M and N to device memor

	hipMalloc(&d_x,x_size*sizeof(float));
	hipMalloc(&d_m, n*sizeof(float));
	hipMalloc(&d_l, n*sizeof(float));
	hipMalloc(&d_I, I_size*sizeof(float));
	hipMalloc(&d_zs, z_size*sizeof(float));
	hipMemcpy(d_x, x_gpu, x_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_m, m_gpu, n*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_I, I_gpu, I_size*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_l, l_gpu, n*sizeof(float), hipMemcpyHostToDevice);
	dim3 dimBlock(6, 6,1);
	dim3 dimGrid(n,1,1);
	Initialize<<<dimGrid, dimBlock>>>(d_x, d_m, d_l, d_I, d_zs, n);	
	hipDeviceSynchronize();
	hipMemcpy(zs_gpu, d_zs, z_size*sizeof(float), hipMemcpyDeviceToHost);
	for(int i = 0; i<n; i++)
	{
		for(int c = 0; c<6; c++)
		{
			for(int r = 0; r<6; r++)
			{	r11=26*n*r+c+i*26;
				r12=r11+6;
				r13=r12+6;
				r21=r13+1;
				r22=r21+6;
				r23=r22+6;
				newbds[i].z11[r][c]=zs_gpu[r11];
				newbds[i].z12[r][c]=zs_gpu[r12];
				newbds[i].z21[r][c]=zs_gpu[r21];
				newbds[i].z22[r][c]=zs_gpu[r22];
				if(c==0)
				{
					newbds[i].z13[r]=zs_gpu[r13];
					newbds[i].z23[r]=zs_gpu[r23];
				}
			}
		}


	}
	
	hipFree(d_x);
	hipFree(d_m);
	hipFree(d_l);
	hipFree(d_I);
	hipFree(d_zs);
	delete[] x_gpu;
	delete[] m_gpu;
	delete[] l_gpu;
	delete[] I_gpu;
	delete[] zs_gpu;


}
