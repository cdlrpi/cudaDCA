#include "hip/hip_runtime.h"
//
//Cuda DCA Driver
//
//So the next thing to do is to load m_gpu, l_gpu, and I_gpu onto the gpu.  
//Then I write something to get initialize ready and interperate the output
//and I check if it matches the initialization pycuda function.
//Then I do the same for Assembly and see if it matches the final assembly
//values from pycuda.  Then we try disassembly and compare again.  Then I 
//get the final acceleration values and check that those are correct.  And
//then I can finally try out the integrator and hope the whole thing works!
#include <malloc.h>
#include "h_code/MainHeader.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <fstream>
//Main function
int main()
{
	//Variable Declarations
	InitBody* bodies;
	Joint* joints;
	float *inits;
	float *Time;
	float *Y;
	
	//System Setup
	int n=3;
	bodies = new InitBody[n];
	joints = new Joint[n];
	inits = new float[2*n];

	//Time Setup
	float tstep=.5;
	float tfinal = 1;
	int tlen = (int) floor(tfinal/tstep);
	Time = new float[tlen];

	//Allocation of memory for the matrix holding the solution 
	Y = new float[2*n];
	//Initialize lengths and masses to 1
	pend_init(bodies,n,1.0,1.0);
	
	//Initialize the position and velocity to match dropping an n link pendulum 
	//where all links are parallel to the horizontal and velocity is 0
	full_drop(inits,n);
	
	//Initialize the time array
	Time_init(Time,tstep,tfinal);

	std::ofstream outfile;
  	outfile.open ("out.txt");
	
  	outfile << "#Time\t";
	
	for(int i=1; i<=n; i++)
	{
		outfile<< "q"<<n<<"\t";
	}
	for(int i =1; i<=n; i++)
	{
		outfile<<"qdot"<<n<<"\t";
	}
	outfile<<"\n";
	for(int i = 0; i< 2*n; i++)
	{
		outfile<<inits[i]<<"\t";
	}
	outfile<<"\n";

	for(int t=0; t<tlen; t++)
	{
		RK_45(inits,tstep,n,bodies,joints,Y);
		for(int i = 0; i<2*n;i++)
		{
			
			outfile<<Y[i]<<"\t";
			inits[i]=Y[i];
		}
		outfile<<"\n";
		
	}
	
	//DCAhelp(inits, bodies, joints,n, Y);
	delete[] inits;
	delete[] Time;
	delete[] Y;
	delete[] bodies;
	delete[] joints;
    outfile.close();
	return 0;

}


