
#include <hip/hip_runtime.h>
#include <iostream>
__global__ void Disassemble_gpu(double Xinv[],double Zs[],double oldAF[], double newAF[], int numBlocks, int lesslen);
void printm(double A[6][6]);
void printa(double A[], int x);


void cudaDisassemble(double OldAF[], double Zs[], double Xs[],double nZs[], double nXs[], int odd, int morelen, int lesslen, double AF[], float times[])
{	
	double *d_oldAF, *d_newAF,*d_Xinv;
	double *d_zs;
	float time1; 
	float time2;

	hipEvent_t beginEvent1;
	hipEvent_t endEvent1;
	hipEvent_t beginEvent2;
	hipEvent_t endEvent2;
	
	int newlen = (int) morelen*4;
	int numBlocks = (int) morelen-lesslen;
 
 	hipEventCreate( &beginEvent1 );
	hipEventCreate( &endEvent1 );
	hipEventRecord( beginEvent1, 0 );
	hipMalloc(&d_zs,sizeof(double)*(morelen)*6*26);
	hipMalloc(&d_newAF,sizeof(double)*(newlen)*6);
	hipMalloc(&d_oldAF,sizeof(double)*(lesslen)*4*6);
	hipMalloc(&d_Xinv,sizeof(double)*(lesslen)*5*5);	
	hipMemcpy(d_zs, Zs, sizeof(double)*(morelen)*6*26, hipMemcpyHostToDevice);
	hipMemcpy(d_Xinv, nXs, sizeof(double)*(lesslen)*5*5, hipMemcpyHostToDevice);
	hipMemcpy(d_oldAF, OldAF, sizeof(double)*(lesslen)*4*6, hipMemcpyHostToDevice);
	hipEventRecord( endEvent1, 0 );
	hipEventSynchronize( endEvent1 );
	hipEventElapsedTime( &time1, beginEvent1, endEvent1 );

	dim3 dimBlock(6, 6,1);
	dim3 dimGrid(numBlocks,1,1);
	
	Disassemble_gpu<<<dimGrid, dimBlock>>>(d_Xinv, d_zs, d_oldAF, d_newAF, morelen,lesslen);
	hipEventCreate( &beginEvent2 );
	hipEventCreate( &endEvent2 );
	hipEventRecord( beginEvent2, 0 );
	hipMemcpy(AF, d_newAF,sizeof(double)*(newlen)*6, hipMemcpyDeviceToHost);
	hipEventRecord( endEvent2, 0 );
	hipEventSynchronize( endEvent2 );
	hipEventElapsedTime( &time2, beginEvent2, endEvent2 );
	if(odd ==1)
	{
		for (int r = 0; r<6;r++)
		{
			AF[r*morelen*4+morelen*4-4]=OldAF[r*lesslen*4+lesslen*4-4];
			AF[r*morelen*4+morelen*4-3]=OldAF[r*lesslen*4+lesslen*4-3];
			AF[r*morelen*4+morelen*4-2]=OldAF[r*lesslen*4+lesslen*4-2];
			AF[r*morelen*4+morelen*4-1]=OldAF[r*lesslen*4+lesslen*4-1];
		}
	}
	
	//std::cin.get();
	times[0] += time1+time2;
	hipFree(d_zs);
	hipFree(d_newAF);
	hipFree(d_oldAF);
	hipFree(d_Xinv);
	
	

}

	

