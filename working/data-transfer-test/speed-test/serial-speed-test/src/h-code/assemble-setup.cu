

#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
__global__ void Assemble_gpu(double oldZs[],double newZs[],double Xinvs[], int nn,int num);
void printm(double A[6][6]);
void printa(double A[], int x);


void cudaAssemble(double Zs[],double Xs[], int num, double nZs[], double nXs[], int odd, int newlen, float times[])
{	
	double *d_ozs,*d_nzs,*d_Xinv;
	float time1; 
	float time2;

	hipEvent_t beginEvent1;
	hipEvent_t endEvent1;
	hipEvent_t beginEvent2;
	hipEvent_t endEvent2;


	

	


	int gpulen=num-odd;
	int numBlocks = (int) (gpulen/2);
	
	hipEventCreate( &beginEvent1 );
	hipEventCreate( &endEvent1 );
	hipEventRecord( beginEvent1, 0 );
	hipMalloc(&d_ozs,sizeof(double)*(num)*6*26);
	hipMalloc(&d_nzs,sizeof(double)*(newlen)*6*26);
	hipMalloc(&d_Xinv,sizeof(double)*(newlen)*5*5);	
	hipMemcpy(d_ozs, Zs, sizeof(double)*(num)*6*26, hipMemcpyHostToDevice);
	hipEventRecord( endEvent1, 0 );
	hipEventSynchronize( endEvent1 );
	hipEventElapsedTime( &time1, beginEvent1, endEvent1 );

	dim3 dimBlock(6, 6,1);
	dim3 dimGrid(numBlocks,1,1);
	hipDeviceSynchronize();
	Assemble_gpu<<<dimGrid, dimBlock>>>(d_ozs, d_nzs, d_Xinv, newlen,num);	
	hipDeviceSynchronize();


	hipEventCreate( &beginEvent2 );
	hipEventCreate( &endEvent2 );
	hipEventRecord( beginEvent2, 0 );
	hipMemcpy(nZs, d_nzs,sizeof(double)*(newlen)*6*26, hipMemcpyDeviceToHost);
	hipMemcpy(nXs, d_Xinv,sizeof(double)*(newlen)*5*5, hipMemcpyDeviceToHost);
	hipEventRecord( endEvent2, 0 );
	hipEventSynchronize( endEvent2 );
	hipEventElapsedTime( &time2, beginEvent2, endEvent2 );
	
	
	if (odd==1)
	{
		for(int r = 0; r<6; r++)
		{
			for(int c=0; c<6; c++)
			{
				nZs[c+r*newlen*26+(newlen-1)*26]=Zs[c+r*num*26+(num-1)*26];	//z11
				nZs[c+r*newlen*26+(newlen-1)*26+6]=Zs[c+r*num*26+(num-1)*26+6];	//z12
				nZs[c+r*newlen*26+(newlen-1)*26+13]=Zs[c+r*num*26+(num-1)*26+13];	//z21
				nZs[c+r*newlen*26+(newlen-1)*26+19]=Zs[c+r*num*26+(num-1)*26+19];	//z22				
				if(r != 5 && c!=5)
				{
					nXs[c+r*newlen*5+(newlen-1)*5]=Xs[c+r*num*5+(num-1)*5];
				}
			}
			nZs[r*newlen*26+(newlen-1)*26+12]=Zs[r*num*26+(num-1)*26+12];	//z13
			nZs[r*newlen*26+(newlen-1)*26+25]=Zs[r*num*26+(num-1)*26+25];	//z23
		}
	}
	//printa(nZs, newlen*6*26);
	
	times[0] += time1+time2;
	hipFree(d_ozs);
	hipFree(d_nzs);
	hipFree(d_Xinv);

}

