#include "hip/hip_runtime.h"
#include "../funct_bin/classes.h"
__global__ void Initialize_gpu(double state[],double m[], double l[],double I[],double Zetas[],int n);
void printa(double a[], int n);
#include <iostream>
#include <stdio.h>
void CudaInitialize(double m[], double l[], double I[], double x[], int n, double Zs[])
{
	int x_size = 2*n;
	int I_size = 3*n*3;
	int z_size = 26*6*n;
	double *d_x, *d_m, *d_l, *d_I, *d_zs;
	
	// Allocate and Load M and N to device memor

	hipMalloc(&d_x,x_size*sizeof(double));
	hipMalloc(&d_m, n*sizeof(double));
	hipMalloc(&d_l, n*sizeof(double));
	hipMalloc(&d_I, I_size*sizeof(double));
	hipMalloc(&d_zs, z_size*sizeof(double));
	hipMemcpy(d_x, x, x_size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_m, m, n*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_I, I, I_size*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_l, l, n*sizeof(double), hipMemcpyHostToDevice);
	dim3 dimBlock(6, 6,1);
	dim3 dimGrid(n,1,1);

	Initialize_gpu<<<dimGrid, dimBlock>>>(d_x, d_m, d_l, d_I, d_zs, n);
	hipMemcpy(Zs, d_zs, z_size*sizeof(double), hipMemcpyDeviceToHost);
	
	hipFree(d_x);
	hipFree(d_m);
	hipFree(d_l);
	hipFree(d_I);
	hipFree(d_zs);
}
