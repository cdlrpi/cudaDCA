#include "hip/hip_runtime.h"
//
//	CUDA DCA Driver
//
//This file invokes all of the necessary function calls to prepare
//and simulate a compound pendulum system through the use of the
//recursive DCA algorithm.	The majority of this algorithm is run
//on the gpu.  Output is created in a format that is
//readable in python for answer checking and graphing purposes.

//Included Files
#include <malloc.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include "h_code/classes.h"
#include "d_code/deviceDisassemble.h"
#include "d_code/deviceAssemble.h"
#include "d_code/deviceInitialize.h"
#include "d_code/deviceFuncts.h"
#include "h_code/npy.h"
#include <math.h>
#include <fstream>
#include <limits>


//Function Prototypes
//	Function found in RK45.cu
void RK_45(double state[], double step, int n, InitBody *bs, Joint *js,double Y[]);

//	Functions found in Functs.cu
void pend_init(InitBody *bs,int n,double mass, double length);
void horizontal_drop(double x[],int n);
void set_up(double A[], double B[], double C[], int n , double h);

//Main function
int main()
{
	int	n=30000;
	while(1)
	{
	n+=10000;
	//Variable Declarations
	InitBody* bodies; //List of bodies used for initialization only
	Joint* joints;  //List of joints between bodies NOTE: This joint list is not used in this version
	double *inits;	//Initial conditions
	double *Y;	//Solution to each timestep
	std::ofstream myfile;
	std::ofstream myfile2;
	myfile2.open("Vals.mtx");
  	myfile.open ("output.mtx");
	//System Setup
	
	bodies = new InitBody[n]; //List of initialization bodies is length n
	joints = new Joint[n]; //List of joints is length n
	inits = new double[2*n];	//Initial conditions are length 2*n
	Y = new double[2*n];	//Timestep solution is length 2*n
	pend_init(bodies,n,1.0,1.0); //Initialize mass, length, and inertia of all bodies 

	//Time Setup
	double tstep= 0.001; //Length of a timestep [s]
	double tfinal =0.003; //Final time [s]
	int tlen = (int) floor(tfinal/tstep)+1;	//Number of timesteps

	//Matrix Output Setup
	//int shape1[2] = { tlen , 2*n }, fortran_order = 0;	//Shape of solution matrix
	//int shape2[2] = { 2 , n+1 };	//Shape of matrix holding information to calculate the energy
	//double Vals[2][n+1];	//Matrix holding information to calculate the energy
	

	typedef std::numeric_limits< double > dbl;

	
	std::cout.precision(dbl::digits10);
	//myfile2<<tstep<<"  ";
	//Vals[0][0]=tstep;	//Save the length of a timestep for plotting purposes	
	//Vals[1][0]=tfinal;	//Save the final time for plotting purposes
	hipEvent_t beginEvent;
	hipEvent_t endEvent;
	hipEventCreate( &beginEvent );
	hipEventCreate( &endEvent );
	for(int i =1; i<n+1; i++)	//Loop through all of the bodies
	{
		myfile2<<bodies[i-1].m<<"  ";
		//Vals[0][i]=bodies[i-1].m;	//Save the mass of body[i]
		//Vals[1][i]=bodies[i-1].l;	//Save the length of body[i]
	}
	//myfile2<<"\n"<<tfinal<<"  ";
	for(int i =1; i<n+1; i++)	//Loop through all of the bodies
	{
		//myfile2<<bodies[i-1].l<<"  ";
		//Vals[0][i]=bodies[i-1].m;	//Save the mass of body[i]
		//Vals[1][i]=bodies[i-1].l;	//Save the length of body[i]
	}
	//System Initialization
	horizontal_drop(inits,n);	//Set the initial conditions
	
	//Save the initial conditions in the solution matrix
	for(int r=0; r<2*n; r++)
	{
		//myfile << inits[r]<< "  ";
	}
	//myfile << "\n";
	hipEventRecord( beginEvent, 0 );
	//Numerical Integration
	for(int t=1; t<tlen; t++)	//Loop through every timestep
	{
		RK_45(inits,tstep,n,bodies,joints,Y);	//Find the solution at that timestep
		for(int i = 0; i<2*n;i++)	//Loop through the solution
		{
			
			inits[i]=Y[i];	//Use the solution as the initial conditions for the next timestep
			//myfile << inits[i]<<"  ";
		}
		//myfile << "\n";
		
	}
	//hipEventRecord( endEvent, 0 );
	//hipEventSynchronize( endEvent );
	float timeValue;
	//hipEventElapsedTime( &timeValue, beginEvent, endEvent );
 	if ( hipSuccess != hipGetLastError() )
    printf( "Error!\n" );
	//std::cout << "Time: " << timeValue << std::endl;
	//Solution Output
	//npy_save_double("Vals.npy",fortran_order,2,shape2,&Vals[0][0]);	//Output values to find energy
	
	//Free memory
	delete[] inits;
	delete[] Y;
	delete[] bodies;
	delete[] joints;
	myfile.close();
	myfile2.close();
	std::cout<<n<<std::endl;
	}
	return EXIT_SUCCESS;	//Program completed successfully
}
