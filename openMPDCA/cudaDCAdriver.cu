#include "hip/hip_runtime.h"
//
//	CUDA DCA Driver
//
//This file invokes all of the necessary function calls to prepare
//and simulate a compound pendulum system through the use of the
//recursive DCA algorithm.	The majority of this algorithm is run
//on the gpu.  Output is created in a format that is
//readable in python for answer checking and graphing purposes.

//Included Files
#include <malloc.h>
#include <iostream>
#include <math.h>
#include "funct_bin/classes.h"
#include "d_code/deviceDisassemble.h"
#include "d_code/deviceAssemble.h"
#include "d_code/deviceInitialize.h"
#include "d_code/deviceFuncts.h"
#include "funct_bin/npy.h"
#include <math.h>
#include <fstream>
#include <limits>


//Function Prototypes
//	Function found in RK45.cu
void RK_45(double state[], double step, int n, double m[], double l[], double II[],double Y[],int cut_off, double Zs[], float times[], int reps);

//	Functions found in Functs.cu
void pend_init(double m[], double l[], double II[],int n,double mass, double length);
void horizontal_drop(double x[],int n);
void set_up(double A[], double B[], double C[], int n , double h);
void Initialize(double m[], double l[],double II[],double Zetas[],int n);

//Main function
int main()
{
	int reps=200;
	int	n=0;
	int cut_off;
	
	std::ofstream timedata;
	std::ofstream numbods;
	numbods.open("numbodsomphyb750.mtx");
	timedata.open("graphomphyb750.mtx");

	int numa;
	for(int xx = 0; xx<4; xx+=1)
	{
		if(xx ==0)
		{
			numa = 0;
		}
		if(xx == 1)
		{
			numa = 1;
		}
		if (xx ==2)
		{
			numa = 3;
		}
		if(xx ==3)
		{
			numa = 6;
		}
	n=0;
	std::cout<<"\n\n\n\n\n"<<numa<<"\n\n\n\n";
	while(n<750)
	{
		if(n<500)
		{
			n+=5;
			
		}
		else if( n<2000)
		{
			n+=20;
		

		}
		else if(n< 10000)
		{
			n+= 100;
		
		}
		else
		{
			n+=10000;
		}

//n+=5;
float *times=(float*)malloc(sizeof(float)*reps);
	int x = n;
	cut_off=x;
	for(int c =0; c<numa; c++)
	{
		if(x==1)
		{
			cut_off=0;
		}
		else if( x%2==0)
		{
			x=x/2;
		}
		else
		{
			x++;
			x=x/2;
		}
	}
	//std::cout<<x<<std::endl;
	if(cut_off !=0)
	{
		cut_off =x;
	}
	

	std::cout<<"xxxx"<<cut_off<<"xxxx";
	//Variable Declarations
	//Variable Declarations
	double *inits;	//Initial conditions
	double *Y;	//Solution to each timestep
	double *m;
	double *l;
	double *II;  //List of joints between bodies NOTE: This joint list is not used in this version
	double *Zs;

	//std::ofstream myfile;
	//std::ofstream myfile2;
	//myfile2.open("Vals.mtx");
  	//myfile.open ("output.mtx");
	//System Setup
	inits = (double*)malloc(sizeof(double)*2*n);	//Initial conditions are length 2*n
	Y = (double*)malloc(sizeof(double)*2*n);	//Timestep solution is length 2*n
	m = (double*)malloc(sizeof(double)*n);
	l = (double*)malloc(sizeof(double)*n);
	II = (double*)malloc(sizeof(double)*n*3*3);
	Zs = (double*)malloc(sizeof(double)*n*26*6);
	pend_init(m,l,II,n,1.0,1.0); //Initialize mass, length, and inertia of all bodies 

	Initialize(m, l, II, Zs, n);

	/////////////////////////////////////////////////////
	


	//Time Setup
	double tstep= 0.001; //Length of a timestep [s]
	double tfinal = 0.001; //Final time [s]
	int tlen = (int) floor(tfinal/tstep)+1;	//Number of timesteps

	//Matrix Output Setup
	//int shape1[2] = { tlen , 2*n }, fortran_order = 0;	//Shape of solution matrix
	//int shape2[2] = { 2 , n+1 };	//Shape of matrix holding information to calculate the energy
	//double Vals[2][n+1];	//Matrix holding information to calculate the energy
	

	typedef std::numeric_limits< double > dbl;

	
	std::cout.precision(dbl::digits10);
	//myfile2<<tstep<<"  ";
	//Vals[0][0]=tstep;	//Save the length of a timestep for plotting purposes	
	//Vals[1][0]=tfinal;	//Save the final time for plotting purposes


	//System Initialization
	horizontal_drop(inits,n);	//Set the initial conditions
	
	//Save the initial conditions in the solution matrix

	//myfile << "\n";

	//Numerical Integration
	//for(int t=1; t<tlen; t++)	//Loop through every timestep
	//{
RK_45(inits,tstep,n,m,l,II,Y,cut_off,Zs,times, reps);	//Find the solution at that timestep

	//	for(int i = 0; i<2*n;i++)	//Loop through the solution
		//{
			
		//	inits[i]=Y[i];	//Use the solution as the initial conditions for the next timestep
			//myfile << inits[i]<<"  ";
	//	}
		//myfile << "\n";
		
//	}
	float timeValue=0;
	for(int i =0; i<reps; i++)
	{
		timeValue+=times[i];
	}
	timeValue/=reps;
	timedata<< timeValue << "  ";
	numbods<<n<<"  ";
 	if ( hipSuccess != hipGetLastError() )
    printf( "Error!\n" );
	std::cout << n << std::endl;
	//Solution Output
	//npy_save_double("Vals.npy",fortran_order,2,shape2,&Vals[0][0]);	//Output values to find energy
	
	//Free memory
free(inits);
	free(II);
	free(m);
	free(l);
	free(Y);
	
	//myfile.close();
	//myfile2.close();
	//std::cout<<n<<std::endl;

	}
	timedata<<"\n";
	numbods<<"\n";

	}
	
	numbods.close();
	timedata.close();
	return EXIT_SUCCESS;	//Program completed successfully
}	
